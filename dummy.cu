#include "dummy.h"
#include <hip/hip_runtime.h>

#define N 32

void cuda_foo()
{
    float buffer[N];
    float *buffer_dev;
    hipMalloc((void **)&buffer_dev, N * sizeof(float));
    hipMemcpy(buffer_dev, buffer, N * sizeof(float), hipMemcpyHostToDevice);
    hipFree(&buffer_dev);
}

#undef N
